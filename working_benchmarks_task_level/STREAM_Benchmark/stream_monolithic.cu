/*
  STREAM benchmark implementation in CUDA.

    COPY:       a(i) = b(i)                 
    SCALE:      a(i) = q*b(i)               
    SUM:        a(i) = b(i) + c(i)          
    TRIAD:      a(i) = b(i) + q*c(i)        

  It measures the memory system on the device.
  The implementation is in double precision.

  Code based on the code developed by John D. McCalpin
  http://www.cs.virginia.edu/stream/FTP/Code/stream.c

  Written by: Massimiliano Fatica, NVIDIA Corporation

  Further modifications by: Ben Cumming, CSCS; Andreas Herten (JSC/FZJ)
*/

//#define NTIMES  20


#include <hip/hip_runtime.h>
#include <string>
#include <vector>

#include <iostream>

#include <stdio.h>
#include <float.h>
#include <limits.h>
#include <unistd.h>
#include <sys/time.h>

#include <sys/time.h>

# ifndef MIN
# define MIN(x,y) ((x)<(y)?(x):(y))
# endif
# ifndef MAX
# define MAX(x,y) ((x)>(y)?(x):(y))
# endif

typedef double real;

static double   avgtime[4] = {0}, maxtime[4] = {0},
        mintime[4] = {FLT_MAX,FLT_MAX,FLT_MAX,FLT_MAX};


void print_help()
{
    printf(
        "Usage: stream [-s] [-n <elements>] [-b <blocksize>] [-t <NTIMES>]\n\n"
        "  -s\n"
        "        Print results in SI units (by default IEC units are used)\n\n"
        "  -n <elements>\n"
        "        Put <elements> values in the arrays\n"
        "        (defaults to 1<<26)\n\n"
        "  -b <blocksize>\n"
        "        Use <blocksize> as the number of threads in each block\n"
        "        (defaults to 192)\n"
        "  -t <NTIMES>\n"
        "        Use NTIMES for the number of times to run the main loop\n"
        "        (defaults to 20)\n"
    );
}

void parse_options(int argc, char** argv, bool& SI, int& N, int& blockSize, int& NTIMES)
{
    // Default values
    SI = false;
    //N = 1<<26;
    N = 100000;
    blockSize = 192;
    NTIMES = 20;

    int c;

    while ((c = getopt (argc, argv, "sn:t:b:h")) != -1)
        switch (c)
        {
            case 's':
                SI = true;
                break;
            case 'n':
                N = std::atoi(optarg);
                break;
            case 'b':
                blockSize = std::atoi(optarg);
                break;
            case 'h':
                print_help();
                std::exit(0);
                break;
            case 't':
                NTIMES = std::atoi(optarg);
                break;
            default:
                print_help();
                std::exit(1);
        }
}

/* A gettimeofday routine to give access to the wall
   clock timer on most UNIX-like systems.  */


double mysecond()
{
    struct timeval tp;
    struct timezone tzp;
    int i = gettimeofday(&tp,&tzp);
    return ( (double) tp.tv_sec + (double) tp.tv_usec * 1.e-6 );
}


template <typename T>
__global__ void set_array(T * __restrict__ const a, T value, int len)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < len)
        a[idx] = value;
}

template <typename T>
__global__ void STREAM_Copy(T const * __restrict__ const a, T * __restrict__ const b, int len)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < len)
        b[idx] = a[idx];
}

template <typename T>
__global__ void STREAM_Scale(T const * __restrict__ const a, T * __restrict__ const b, T scale,  int len)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < len)
        b[idx] = scale * a[idx];
}

template <typename T>
__global__ void STREAM_Add(T const * __restrict__ const a, T const * __restrict__ const b, T * __restrict__ const c, int len)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < len)
        c[idx] = a[idx] + b[idx];
}

template <typename T>
__global__ void STREAM_Triad(T const * __restrict__ a, T const * __restrict__ b, T * __restrict__ const c, T scalar, int len)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < len)
        c[idx] = a[idx] + scalar * b[idx];
}

template <typename T>
__global__ void stream_monolithic(T const * __restrict__ const a, T * __restrict__ const b,  T * __restrict__ const c, T scale,  int len)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < len) {
        // Copy
        b[idx] = a[idx];

        // Scale
        b[idx] = scale * a[idx];

        // Add
        c[idx] = a[idx] + b[idx];

        // Triad
        c[idx] = a[idx] + scale * b[idx];
    }
}

int main(int argc, char** argv)
{
    real *d_a, *d_b, *d_c;
    int j,k;
    real scalar;
    std::vector<std::string> label{"Copy:      ", "Scale:     ", "Add:       ", "Triad:     "};

    // Parse arguments
    bool SI;
    int N, blockSize, NTIMES;
    parse_options(argc, argv, SI, N, blockSize, NTIMES);

    printf(" STREAM Benchmark implementation in CUDA\n");
    printf(" Array size (%s precision) =%7.2f MB\n", sizeof(double)==sizeof(real)?"double":"single", double(N)*double(sizeof(real))/1.e6);

    /* Allocate memory on device */
    hipMalloc((void**)&d_a, sizeof(real)*N);
    hipMalloc((void**)&d_b, sizeof(real)*N);
    hipMalloc((void**)&d_c, sizeof(real)*N);

    /* Compute execution configuration */
    dim3 dimBlock(blockSize);
    dim3 dimGrid(N/dimBlock.x );
    if( N % dimBlock.x != 0 ) dimGrid.x+=1;

    printf(" using %d threads per block, %d blocks\n",dimBlock.x,dimGrid.x);

    if (SI)
        printf(" output in SI units (KB = 1000 B)\n");
    else
        printf(" output in IEC units (KiB = 1024 B)\n");

    /* Initialize memory on the device */
    set_array<real><<<dimGrid,dimBlock>>>(d_a, 2.f, N);
    set_array<real><<<dimGrid,dimBlock>>>(d_b, .5f, N);
    set_array<real><<<dimGrid,dimBlock>>>(d_c, .5f, N);

    /*  --- MAIN LOOP --- repeat test cases NTIMES times --- */

    float milliseconds = 0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    scalar=3.0f;
    for (k=0; k<NTIMES; k++)
    {
        stream_monolithic<real><<<dimGrid,dimBlock>>>(d_a, d_b, d_c, scalar, N);
    }

    hipEventRecord(stop);

    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    std::cout << "Total Time: " << milliseconds << " ms" << std::endl;

    /* Free memory on device */
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}
