#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <chrono>
#include <hip/hip_runtime.h>
#include "mv.h"

// sparse matrix vector multiply using the CSR format
__global__ void mv_csr_split_atomic(const int num_rows,
                       const size_t *row_indices,
                       const int *col_indices,
                       const REAL *values,
                       const REAL *x,
                             REAL *y,
                       const int stream_A,
                       const int num_streams)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < num_rows) {
    size_t row_start = row_indices[i];
    size_t row_end = row_indices[i+1];

    if (stream_A == 0)
    {
      y[i] = 0;
    }

    REAL temp = 0;

      // 나눠진 index 범위 정의
    const size_t split_start = stream_A * (row_end - row_start) / num_streams + row_start;
    const size_t split_end   = stream_A + 1 == num_streams ? row_end : (stream_A + 1) * (row_end - row_start) / num_streams + row_start;

    for(size_t n = split_start; n < split_end; n++){
      temp += values[n] * x[col_indices[n]];
    }

    atomicAdd(&y[i], temp);
    /*
    if (stream_A) {
      y[i] = temp;
    } else {
      y[i] += temp;
    }
    */

  }
}

// dense matrix vector multiply
__global__ void mv_dense(const int num_rows, const REAL* matrix, const REAL* x, REAL* y)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < num_rows) {
    REAL temp = 0;
    for (int j = 0; j < num_rows; j++) {
      if (matrix[i * num_rows + j] != (REAL)0) 
        temp += matrix[i * num_rows + j] * x[j];
    }
    y[i] = temp;
  }
}

long mv_dense_parallel(const int repeat,
                       const int bs,
                       const int num_rows,
                       const REAL* x,
                             REAL* matrix,
                             REAL* y)
{
  REAL *d_x, *d_matrix, *d_y;
  hipMalloc(&d_x, num_rows*sizeof(REAL));
  hipMalloc(&d_matrix, num_rows * num_rows * sizeof(REAL));
  hipMalloc(&d_y, num_rows*sizeof(REAL));

  hipMemcpy(d_x, x, num_rows*sizeof(REAL), hipMemcpyHostToDevice);
  hipMemcpy(d_matrix, matrix, num_rows*num_rows*sizeof(REAL), hipMemcpyHostToDevice);
  

  dim3 grids ((num_rows + bs - 1) / bs);
  dim3 blocks (bs);

  hipDeviceSynchronize();
  auto start=std::chrono::steady_clock::now();
MY_START_CLOCK(cuda simpleSpmv-cuda kernels.cu,0);

  for (int i = 0; i < repeat; i++)
    mv_dense<<<grids, blocks>>>(num_rows, d_matrix, d_x, d_y);

  hipDeviceSynchronize();
  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono:: duration_cast<std::chrono::nanoseconds>(end - start).count();
MY_STOP_CLOCK(cuda simpleSpmv-cuda kernels.cu,0);
  hipMemcpy(y, d_y, num_rows*sizeof(REAL), hipMemcpyDeviceToHost);

  hipFree(d_x);
  hipFree(d_y);
  hipFree(d_matrix);

  return time;
}

long mv_csr_parallel(const int repeat,
                     const int bs,
                     const int num_rows,
                     const REAL* x,
                     const size_t nnz,
                     REAL* matrix,
                     REAL* y)
{
  size_t *row_indices = (size_t *) malloc((num_rows+1) * sizeof(size_t));
  int *col_indices = (int *) malloc(nnz * sizeof(int));
  REAL *values = (REAL *) malloc(nnz * sizeof(REAL));

  // initialize csr structure
  init_csr(row_indices, values, col_indices, matrix, num_rows, nnz);

  size_t *d_row_indices;
  int *d_col_indices;
  REAL *d_values, *d_x, *d_y;

  hipMalloc(&d_row_indices, (num_rows+1)*sizeof(size_t));
  hipMalloc(&d_col_indices, nnz*sizeof(int));
  hipMalloc(&d_values, nnz*sizeof(REAL));
  hipMalloc(&d_x, num_rows*sizeof(REAL));
  hipMalloc(&d_y, num_rows*sizeof(REAL));

  hipMemcpy(d_row_indices, row_indices, (num_rows+1)*sizeof(size_t), hipMemcpyHostToDevice);
  hipMemcpy(d_col_indices, col_indices, nnz*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_values, values, nnz*sizeof(REAL), hipMemcpyHostToDevice);
  hipMemcpy(d_x, x, num_rows*sizeof(REAL), hipMemcpyHostToDevice);
  //before the computation

  dim3 grids ((num_rows + bs - 1) / bs);
  dim3 blocks (bs);

  hipDeviceSynchronize();
  auto start=std::chrono::steady_clock::now();
  MY_START_CLOCK(cuda simpleSpmv-cuda kernels.cu,1);

  // N-way split define
  int n_way_split = 10;
  hipStream_t* streams = new hipStream_t[n_way_split];
  for (int i = 0; i < n_way_split; ++i) {
    hipStreamCreateWithFlags(&streams[i], hipStreamNonBlocking);
  }

  for (int i = 0; i < repeat; i++)
  {
    hipMemset(d_y, 0, num_rows*sizeof(REAL)); // Initialize y to zero 

    for (int j = 0; j < n_way_split; ++j) {
      mv_csr_split_atomic<<<grids, blocks, 0, streams[j]>>>(num_rows, d_row_indices, d_col_indices, d_values, d_x, d_y, j, n_way_split); // N-way split
      }

      for (int j = 0; j < n_way_split; ++j) {
        hipStreamSynchronize(streams[j]);
      }
  }

  hipDeviceSynchronize();
  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono:: duration_cast<std::chrono::nanoseconds>(end - start).count();
MY_STOP_CLOCK(cuda simpleSpmv-cuda kernels.cu,1);

  hipMemcpy(y, d_y, num_rows*sizeof(REAL), hipMemcpyDeviceToHost);

  free(values);
  free(row_indices);
  free(col_indices);

  hipFree(d_row_indices);
  hipFree(d_col_indices);
  hipFree(d_values);
  hipFree(d_x);
  hipFree(d_y);

  return time;
}
