#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <chrono>
#include <hip/hip_runtime.h>
#include "mv.h"

// sparse matrix vector multiply using the CSR format
__global__ void mv_csr_pipeline(const int num_rows,
                       const size_t *row_indices,
                       const int *col_indices,
                       const REAL *values,
                       const REAL *x,
                             REAL *y,
                       REAL *tmp_saved,
                       const bool stream_A)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < num_rows) {
    size_t row_start = row_indices[i];
    size_t row_end = row_indices[i+1];

    REAL temp = 0;

      // 나눠진 index 범위 정의
    const size_t split_start = stream_A ? row_start : (row_end - row_start) / 2 + row_start;
    const size_t split_end   = stream_A ? (row_end - row_start) / 2 + row_start : row_end;

    for(size_t n = split_start; n < split_end; n++){
      temp += values[n] * x[col_indices[n]];
    }

    if (stream_A) {
      tmp_saved[i] = temp;
    } else {
      y[i] = temp + tmp_saved[i];
    }

  }
}

// dense matrix vector multiply
__global__ void mv_dense(const int num_rows, const REAL* matrix, const REAL* x, REAL* y)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < num_rows) {
    REAL temp = 0;
    for (int j = 0; j < num_rows; j++) {
      if (matrix[i * num_rows + j] != (REAL)0) 
        temp += matrix[i * num_rows + j] * x[j];
    }
    y[i] = temp;
  }
}

long mv_dense_parallel(const int repeat,
                       const int bs,
                       const int num_rows,
                       const REAL* x,
                             REAL* matrix,
                             REAL* y)
{
  REAL *d_x, *d_matrix, *d_y;
  hipMalloc(&d_x, num_rows*sizeof(REAL));
  hipMalloc(&d_matrix, num_rows * num_rows * sizeof(REAL));
  hipMalloc(&d_y, num_rows*sizeof(REAL));

  hipMemcpy(d_x, x, num_rows*sizeof(REAL), hipMemcpyHostToDevice);
  hipMemcpy(d_matrix, matrix, num_rows*num_rows*sizeof(REAL), hipMemcpyHostToDevice);

  dim3 grids ((num_rows + bs - 1) / bs);
  dim3 blocks (bs);

  hipDeviceSynchronize();
  auto start=std::chrono::steady_clock::now();
MY_START_CLOCK(cuda simpleSpmv-cuda kernels.cu,0);

  for (int i = 0; i < repeat; i++)
    mv_dense<<<grids, blocks>>>(num_rows, d_matrix, d_x, d_y);

  hipDeviceSynchronize();
  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono:: duration_cast<std::chrono::nanoseconds>(end - start).count();
MY_STOP_CLOCK(cuda simpleSpmv-cuda kernels.cu,0);
  hipMemcpy(y, d_y, num_rows*sizeof(REAL), hipMemcpyDeviceToHost);

  hipFree(d_x);
  hipFree(d_y);
  hipFree(d_matrix);

  return time;
}

long mv_csr_parallel(const int repeat,
                     const int bs,
                     const int num_rows,
                     const REAL* x,
                     const size_t nnz,
                     REAL* matrix,
                     REAL* y)
{
  size_t *row_indices = (size_t *) malloc((num_rows+1) * sizeof(size_t));
  int *col_indices = (int *) malloc(nnz * sizeof(int));
  REAL *values = (REAL *) malloc(nnz * sizeof(REAL));

  // initialize csr structure
  init_csr(row_indices, values, col_indices, matrix, num_rows, nnz);

  size_t *d_row_indices;
  int *d_col_indices;
  REAL *d_values, *d_x, *d_y;
  REAL *d_tmp_saved;

  hipMalloc(&d_row_indices, (num_rows+1)*sizeof(size_t));
  hipMalloc(&d_col_indices, nnz*sizeof(int));
  hipMalloc(&d_values, nnz*sizeof(REAL));
  hipMalloc(&d_x, num_rows*sizeof(REAL));
  hipMalloc(&d_y, num_rows*sizeof(REAL));
  hipMalloc(&d_tmp_saved, (num_rows)*sizeof(REAL));

  hipMemcpy(d_row_indices, row_indices, (num_rows+1)*sizeof(size_t), hipMemcpyHostToDevice);
  hipMemcpy(d_col_indices, col_indices, nnz*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_values, values, nnz*sizeof(REAL), hipMemcpyHostToDevice);
  hipMemcpy(d_x, x, num_rows*sizeof(REAL), hipMemcpyHostToDevice);
  hipMemset(d_tmp_saved, 0, (num_rows)*sizeof(REAL));

  dim3 grids ((num_rows + bs - 1) / bs);
  dim3 blocks (bs);

  hipDeviceSynchronize();
  auto start=std::chrono::steady_clock::now();
  MY_START_CLOCK(cuda simpleSpmv-cuda kernels.cu,1);

  hipStream_t stream1, stream2;
  hipStreamCreateWithFlags(&stream1, hipStreamNonBlocking);
  hipStreamCreateWithFlags(&stream2, hipStreamNonBlocking);

  hipEvent_t event[(repeat)];  // NUM = number of (A,B) pairs
  for (int i = 0; i < (repeat); ++i) {
      hipEventCreate(&event[i]);
  }

  for (int i = 0; i < repeat; i++)
  {
    mv_csr_pipeline<<<grids, blocks, 0, stream1>>>(num_rows, d_row_indices, d_col_indices, d_values, d_x, d_y, d_tmp_saved, true);

    hipEventRecord(event[i], stream1);
    hipStreamWaitEvent(stream2, event[i], 0);
    mv_csr_pipeline<<<grids, blocks, 0, stream2>>>(num_rows, d_row_indices, d_col_indices, d_values, d_x, d_y, d_tmp_saved, false);
  }

  hipDeviceSynchronize();
  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono:: duration_cast<std::chrono::nanoseconds>(end - start).count();
MY_STOP_CLOCK(cuda simpleSpmv-cuda kernels.cu,1);

  hipMemcpy(y, d_y, num_rows*sizeof(REAL), hipMemcpyDeviceToHost);

  free(values);
  free(row_indices);
  free(col_indices);

  hipFree(d_row_indices);
  hipFree(d_col_indices);
  hipFree(d_values);
  hipFree(d_x);
  hipFree(d_y);

  return time;
}
