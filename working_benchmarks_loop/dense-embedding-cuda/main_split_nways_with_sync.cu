#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <chrono>
#include <random>
#include <hip/hip_runtime.h>

template <typename T>
void reference(
    const T* input,
    const T* dense,
    T* output,
    int embedding_dim,
    int batch_size,
    const int* offset)
{
  for (int batch_idx = 0; batch_idx < batch_size; batch_idx++) {
    const int range = offset[batch_idx + 1] - offset[batch_idx];
    for (int idx = 0; idx < embedding_dim; idx++) {
      const T dense_elem = dense[batch_idx * embedding_dim + idx];
      for (int nested_idx = idx; nested_idx < range; nested_idx += embedding_dim) {
        output[offset[batch_idx] + nested_idx] = 
          input[offset[batch_idx] + nested_idx] + dense_elem;
      }
    }
  }
}

template <typename T>
__global__ void dense_esuhm_split(
    const T* input,
    const T* dense,
          T* output,
    int embedding_dim,
    const int* offset,
    const int stream_A,
    const int num_streams)
{
  const int batch_idx  = blockIdx.x; // each batch is handled by a block
  const int grain_size = blockDim.x;
  const int tid = threadIdx.x;
  const int range = offset[batch_idx + 1] - offset[batch_idx];

  const int split_start = stream_A * (embedding_dim / num_streams);
  const int split_end = stream_A + 1 == num_streams ? embedding_dim : (stream_A + 1)* (embedding_dim / num_streams);

  for (int idx = tid + split_start; idx < split_end; idx += grain_size) {
    const T dense_elem = dense[batch_idx * embedding_dim + idx];
    for (int nested_idx = idx; nested_idx < range; nested_idx += embedding_dim) {
      output[offset[batch_idx] + nested_idx] = input[offset[batch_idx] + nested_idx] + dense_elem;
    }
  }
}

template <typename T>
__global__ void dense_esuhm2_split(
    const T* input,
    const T* dense,
          T* output,
    int embedding_dim,
    const int* offset,
    const int stream_A,
    const int num_streams)
{
  const int batch_idx  = blockIdx.x;
  const int start = offset[batch_idx];
  const int range = offset[batch_idx + 1] - start;

  const int split_start = stream_A * (embedding_dim / num_streams);
  const int split_end = stream_A + 1 == num_streams ? embedding_dim : (stream_A + 1)* (embedding_dim / num_streams);

  for (int idx = threadIdx.x + split_start; idx < split_end; idx += blockDim.x) {
    const T dense_elem = dense[batch_idx * embedding_dim + idx];
    for (int nested_idx = idx; nested_idx < range; nested_idx += embedding_dim) {
      output[start + nested_idx] = input[start + nested_idx] + dense_elem;
    }
  }
}

int main(int argc, char* argv[])
{
  if (argc != 4) {
    printf("Usage: %s <number of rows> <batch size> <repeat>\n", argv[0]);
    return 1;
  }
  const int nrows = atoi(argv[1]);
  const int batch_size = atoi(argv[2]);
  const int repeat = atoi(argv[3]);
  assert(nrows > batch_size * batch_size);

  printf("Number of rows in the embedding table: %d\n", nrows);
  printf("Batch size: %d\n", batch_size);

  for (int ncols = 2048; ncols <= 2048; ncols = ncols * 2) {

    printf("\nEmbedding dimension: %d\n", ncols);

    int input_size = nrows * ncols;  // same as output size
    size_t input_size_bytes = input_size * sizeof(float);

    int dense_size = batch_size * ncols ;
    int dense_size_bytes = dense_size * sizeof(float);

    int batch_size_bytes = (batch_size + 1) * sizeof(float);

    float *input, *dense, *output, *output_ref;
    input = (float*) malloc (input_size_bytes); // [sum(*) x embedding_dim]
    dense = (float*) malloc (dense_size_bytes); // [batch_size x embedding_dim]
    output = (float*) malloc (input_size_bytes); // [sum(*) x embedding_dim]
    output_ref = (float*) malloc (input_size_bytes); // [sum(*) x embedding_dim]
    int *input_offset = (int*) malloc (batch_size_bytes);  // [batch_size]

    // meet a few conditions:
    // range > ncols
    // range + offset[i] < input_size
    // offset[i] > offset[i-1]
    srand(123);
    input_offset[0] = 0;
    for (int i = 1; i <= batch_size; i++)
      input_offset[i] = input_offset[i-1] + (rand() % batch_size + 1) * ncols;

    std::default_random_engine g (123);
    std::uniform_real_distribution<float> distr (-1.f, 1.f);
    for (int i = 0; i < dense_size; i++) {
      dense[i] = distr(g);
    }

    for (int i = 0; i < input_size; i++) {
      input[i] = distr(g);
      output_ref[i] = 0;
    }

    reference(input, dense, output_ref, ncols, batch_size, input_offset);

    float *d_input, *d_dense, *d_output;
    hipMalloc((void**)&d_input, input_size_bytes);
    hipMemcpy(d_input, input, input_size_bytes, hipMemcpyHostToDevice);
    
    hipMalloc((void**)&d_dense, dense_size_bytes);
    hipMemcpy(d_dense, dense, dense_size_bytes, hipMemcpyHostToDevice);

    hipMalloc((void**)&d_output, input_size_bytes);
    hipMemset(d_output, 0, input_size_bytes);

    int* d_input_offset;
    hipMalloc((void**)&d_input_offset, batch_size_bytes);
    hipMemcpy(d_input_offset, input_offset, batch_size_bytes, hipMemcpyHostToDevice);
    
    dim3 grid (batch_size);
    dim3 block (256);

    hipDeviceSynchronize();

    auto start=std::chrono::steady_clock::now();MY_START_CLOCK(cuda dense-embedding-cuda main.cu,0);

    // N-way split define
    int n_way_split = 8;

    hipStream_t* streams = new hipStream_t[n_way_split];
    for (int i = 0; i < n_way_split; ++i) {
      hipStreamCreateWithFlags(&streams[i], hipStreamNonBlocking);
    }

    for (int i = 0; i < repeat; i++) 
    {
      for (int j = 0; j < n_way_split; ++j) {
         dense_esuhm_split<<<grid, block, 0, streams[j]>>>(d_input, d_dense, d_output, ncols, d_input_offset, j, n_way_split);
      }
      for (int j = 0; j < n_way_split; ++j) {
        hipStreamSynchronize(streams[j]);
      }
    }

    hipDeviceSynchronize();
    auto end = std::chrono::steady_clock::now();
    auto time = std::chrono:: duration_cast<std::chrono::nanoseconds>(end - start).count();MY_STOP_CLOCK(cuda dense-embedding-cuda main.cu,0);
    printf("Average execution time of dense embedding kernel (k1): %f (us)\n", (time * 1e-3f) / repeat);

    hipDeviceSynchronize();
    start=std::chrono::steady_clock::now();MY_START_CLOCK(cuda dense-embedding-cuda main.cu,1);

    for (int i = 0; i < repeat; i++) 
    {
      for (int j = 0; j < n_way_split; ++j) {
      dense_esuhm2_split<<<grid, block, 0, streams[j]>>>(d_input, d_dense, d_output, ncols, d_input_offset, j, n_way_split);
      }
    }

    hipDeviceSynchronize();
    end = std::chrono::steady_clock::now();
    time = std::chrono:: duration_cast<std::chrono::nanoseconds>(end - start).count();MY_STOP_CLOCK(cuda dense-embedding-cuda main.cu,1);
    printf("Average execution time of dense embedding kernel (k2): %f (us)\n", (time * 1e-3f) / repeat);

    hipMemcpy(output, d_output, input_size_bytes, hipMemcpyDeviceToHost);

    bool ok = true;
    for (int i = 0; i < input_size; i++) {
      if (fabsf(output[i] - output_ref[i]) > 1e-3f) {
        ok = false;
        break;
      }
    }
    printf("%s\n", ok ? "PASS" : "FAIL");

    hipFree(d_input);
    hipFree(d_dense);
    hipFree(d_output);
    hipFree(d_input_offset);

    free(input);
    free(dense);
    free(output);
    free(output_ref);
    free(input_offset);
  }

  return 0;
}
