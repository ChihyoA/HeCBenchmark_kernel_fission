#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

///////////////////////////////////////////////////////////////////////////////
// This sample implements Niederreiter quasirandom number generator
// and Moro's Inverse Cumulative Normal Distribution generator
///////////////////////////////////////////////////////////////////////////////

// standard utilities and systems includes
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <chrono>
#include <hip/hip_runtime.h>
#include "qrg.h"

// forward declarations
void initQuasirandomGenerator(unsigned int *table);
double getQuasirandomValue63(INT64 i, int dim);
double MoroInvCNDcpu(unsigned int x);

// Round Up Division function
size_t shrRoundUp(int group_size, int global_size) 
{
  int r = global_size % group_size;
  if(r == 0) 
  {
    return global_size;
  } else 
  {
    return global_size + group_size - r;
  }
}

////////////////////////////////////////////////////////////////////////////////
// Moro's Inverse Cumulative Normal Distribution function approximation
////////////////////////////////////////////////////////////////////////////////
__device__
float MoroInvCNDgpu(unsigned int x)
{
  const float a1 = 2.50662823884f;
  const float a2 = -18.61500062529f;
  const float a3 = 41.39119773534f;
  const float a4 = -25.44106049637f;
  const float b1 = -8.4735109309f;
  const float b2 = 23.08336743743f;
  const float b3 = -21.06224101826f;
  const float b4 = 3.13082909833f;
  const float c1 = 0.337475482272615f;
  const float c2 = 0.976169019091719f;
  const float c3 = 0.160797971491821f;
  const float c4 = 2.76438810333863E-02f;
  const float c5 = 3.8405729373609E-03f;
  const float c6 = 3.951896511919E-04f;
  const float c7 = 3.21767881768E-05f;
  const float c8 = 2.888167364E-07f;
  const float c9 = 3.960315187E-07f;

  float z;

  bool negate = false;

  // Ensure the conversion to floating point will give a value in the
  // range (0,0.5] by restricting the input to the bottom half of the
  // input domain. We will later reflect the result if the input was
  // originally in the top half of the input domain
  if (x >= 0x80000000UL)
  {
    x = 0xffffffffUL - x;
    negate = true;
  }

  // x is now in the range [0,0x80000000) (i.e. [0,0x7fffffff])
  // Convert to floating point in (0,0.5]
  const float x1 = 1.0f / (float)0xffffffffUL;
  const float x2 = x1 / 2.0f;
  float p1 = x * x1 + x2;
  // Convert to floating point in (-0.5,0]
  float p2 = p1 - 0.5f;

  // The input to the Moro inversion is p2 which is in the range
  // (-0.5,0]. This means that our output will be the negative side
  // of the bell curve (which we will reflect if "negate" is true).

  // Main body of the bell curve for |p| < 0.42
  if (p2 > -0.42f)
  {
    z = p2 * p2;
    z = p2 * (((a4 * z + a3) * z + a2) * z + a1) / ((((b4 * z + b3) * z + b2) * z + b1) * z + 1.0f);
  }
  // Special case (Chebychev) for tail
  else
  {
    z = logf(-logf(p1));
    z = - (c1 + z * (c2 + z * (c3 + z * (c4 + z * (c5 + z * (c6 + z * (c7 + z * (c8 + z * c9))))))));
  }

  // If the original input (x) was in the top half of the range, reflect
  // to get the positive side of the bell curve
  return negate ? -z : z;
}

// size of output random array
const unsigned int N = 1048576;

__global__ void  
qrng (float* output, const unsigned int* table, const unsigned int seed, const unsigned int N, const int stream_A, const int num_streams)
{
  unsigned int globalID_x   = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int localID_y    = threadIdx.y;
  unsigned int globalSize_x = gridDim.x * blockDim.x;

  // 나눠진 index 범위 정의
  //const size_t split_start = stream_A ? 0 : (N / 2);
  //const size_t split_end   = stream_A ? (N / 2) : N;

  //const int split_start = stream_A * ((N - globalID_x) / num_streams);
  //const int split_end = stream_A + 1 == num_streams ? N : globalID_x + (stream_A + 1)* ((N - globalID_x) / num_streams);

  const int split_start = stream_A * ((N) / num_streams);
  const int split_end = stream_A + 1 == num_streams ? N : (stream_A + 1)* ((N) / num_streams);
  
  for (unsigned int pos = globalID_x+split_start; pos < split_end; pos += globalSize_x) {
    unsigned int result = 0;
    unsigned int data = seed + pos;
    for(int bit = 0; bit < QRNG_RESOLUTION; bit++, data >>= 1)
      if(data & 1) result ^= table[bit+localID_y*QRNG_RESOLUTION];
    output[__mul24(localID_y,N) + pos] = (float)(result + 1) * INT_SCALE;
  }
}

__global__ void  
icnd (float* output, const unsigned int pathN, const unsigned int distance, const int stream_A, const int num_streams)
{
  const unsigned int globalID   = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int globalSize = gridDim.x * blockDim.x;

  // 나눠진 index 범위 정의
  //const size_t split_start = stream_A ? 0 : (pathN / 2);
  //const size_t split_end   = stream_A ? (pathN / 2) : pathN;

  //const int split_start = stream_A * ((pathN - globalID) / num_streams);
  //const int split_end = stream_A + 1 == num_streams ? pathN : globalID + (stream_A + 1)* ((pathN - globalID) / num_streams);

  const int split_start = stream_A * ((pathN) / num_streams);
  const int split_end = stream_A + 1 == num_streams ? pathN : globalID + (stream_A + 1)* ((pathN) / num_streams);


  for(unsigned int pos = globalID+split_start; pos < split_end; pos += globalSize){
    unsigned int d = (pos + 1) * distance;
    output[pos] = MoroInvCNDgpu(d);
  }
}

int main(int argc, const char **argv)
{
  if (argc != 2) {
    printf("Usage: %s <repeat>\n", argv[0]);
    return 1;
  }
  const int repeat = atoi(argv[1]);

  unsigned int dim, pos;
  double delta, ref, sumDelta, sumRef, L1norm;
  unsigned int tableCPU[QRNG_DIMENSIONS*QRNG_RESOLUTION];
  bool bPassFlag = false;

  float* h_OutputGPU = (float *)malloc(QRNG_DIMENSIONS * N * sizeof(float));

  printf("Initializing QRNG tables...\n");
  initQuasirandomGenerator(tableCPU);

  float *d_Output;
  hipMalloc((void**)&d_Output, sizeof(float)*QRNG_DIMENSIONS*N);

  unsigned int* d_Table;
  hipMalloc((void**)&d_Table, sizeof(unsigned int)*QRNG_DIMENSIONS*QRNG_RESOLUTION);
  hipMemcpy(d_Table, tableCPU, sizeof(unsigned int)*QRNG_DIMENSIONS*QRNG_RESOLUTION, 
      hipMemcpyHostToDevice);

  printf(">>>Launch QuasirandomGenerator kernel...\n\n"); 

  size_t szWorkgroup = 64 * (256 / QRNG_DIMENSIONS)/64;
  size_t globalWorkSize[2] = {shrRoundUp(szWorkgroup, 128*128), QRNG_DIMENSIONS};
  size_t localWorkSize[2] = {szWorkgroup, QRNG_DIMENSIONS};
  dim3 grid (globalWorkSize[0] / localWorkSize[0], globalWorkSize[1] / localWorkSize[1]);
  dim3 block (localWorkSize[0], localWorkSize[1]);

  // seed is fixed at zero
  const unsigned int seed = 0;

  hipDeviceSynchronize();
  auto start=std::chrono::steady_clock::now();MY_START_CLOCK(cuda qrg-cuda main.cu,0);

  // N-way split define
  int n_way_split = 6;

  hipStream_t* streams = new hipStream_t[n_way_split];
  for (int i = 0; i < n_way_split; ++i) {
    hipStreamCreateWithFlags(&streams[i], hipStreamNonBlocking);
  }

  for (int i = 0; i < repeat; i++)
  {
    for (int j = 0; j < n_way_split; ++j) {
    qrng<<<grid, block, 0, streams[j]>>> (d_Output, d_Table, seed, N, j, n_way_split);
    
    }

    for (int j = 0; j < n_way_split; ++j) {
      hipStreamSynchronize(streams[j]);
    }
  }

  hipDeviceSynchronize();
  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono:: duration_cast<std::chrono::nanoseconds>(end - start).count();MY_STOP_CLOCK(cuda qrg-cuda main.cu,0);
  printf("Average kernel execution time (qrng): %f (us)\n", (time * 1e-3f) / repeat);

  printf("\nRead back results...\n"); 
  hipMemcpy(h_OutputGPU, d_Output, sizeof(float)*QRNG_DIMENSIONS*N, hipMemcpyDeviceToHost);

  printf("Comparing to the CPU results...\n\n");
  sumDelta = 0;
  sumRef   = 0;
  for(dim = 0; dim < QRNG_DIMENSIONS; dim++)
  {
    for(pos = 0; pos < N; pos++)
    {
      ref       = getQuasirandomValue63(pos, dim);
      delta     = (double)h_OutputGPU[dim * N  + pos] - ref;
      sumDelta += fabs(delta);
      sumRef   += fabs(ref);
    }
  }
  L1norm = sumDelta / sumRef;
  printf("  L1 norm: %E\n", L1norm);
  printf("  ckQuasirandomGenerator deviations %s Allowable Tolerance\n\n\n", (L1norm < 1e-6) ? "WITHIN" : "ABOVE");
  bPassFlag = (L1norm < 1e-6);

  printf(">>>Launch InverseCND kernel...\n\n"); 

  // reuse variables for work-group sizes
  szWorkgroup = 128;
  globalWorkSize[0] = shrRoundUp(szWorkgroup, 128*128);
  localWorkSize[0] = szWorkgroup;

  dim3 grid2 (globalWorkSize[0] / localWorkSize[0]);
  dim3 block2 (localWorkSize[0]);

  const unsigned int pathN = QRNG_DIMENSIONS * N;
  const unsigned int distance = ((unsigned int)-1) / (pathN  + 1);

  hipDeviceSynchronize();
  start=std::chrono::steady_clock::now();MY_START_CLOCK(cuda qrg-cuda main.cu,1);

  for (int i = 0; i < repeat; i++)
  {
    for (int j = 0; j < n_way_split; ++j) {
    icnd<<<grid2, block2, 0, streams[j]>>>(d_Output, pathN, distance, j, n_way_split);
    }

    for (int j = 0; j < n_way_split; ++j) {
      hipStreamSynchronize(streams[j]);
    }
  }

  hipDeviceSynchronize();
  end = std::chrono::steady_clock::now();
  time = std::chrono:: duration_cast<std::chrono::nanoseconds>(end - start).count();MY_STOP_CLOCK(cuda qrg-cuda main.cu,1);
  printf("Average kernel execution time (icnd): %f (us)\n", (time * 1e-3f) / repeat);

  printf("\nRead back results...\n"); 
  hipMemcpy(h_OutputGPU, d_Output, sizeof(float)*QRNG_DIMENSIONS*N, hipMemcpyDeviceToHost);

  printf("Comparing to the CPU results...\n\n");
  sumDelta = 0;
  sumRef   = 0;
  for(pos = 0; pos < QRNG_DIMENSIONS * N; pos++){
    unsigned int d = (pos + 1) * distance;
    ref       = MoroInvCNDcpu(d);
    delta     = (double)h_OutputGPU[pos] - ref;
    sumDelta += fabs(delta);
    sumRef   += fabs(ref);
  }
  L1norm = sumDelta / sumRef;
  printf("  L1 norm: %E\n", L1norm);
  printf("  ckInverseCNDGPU deviations %s Allowable Tolerance\n\n\n", (L1norm < 1e-6) ? "WITHIN" : "ABOVE");
  bPassFlag &= (L1norm < 1e-6);

  if (bPassFlag)
    printf("PASS\n");
  else
    printf("FAIL\n");

  free(h_OutputGPU);
  hipFree(d_Output);
  hipFree(d_Table);
  return 0;
}
